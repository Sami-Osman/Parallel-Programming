//Function decleration
hipError_t mergeSortWithCuda(int* arr, unsigned int count, unsigned int kernelBlockSize, bool analysis_flag, bool SHEMEM_flag);

hipError_t mergeSortWithCuda(int* arr, unsigned int count, unsigned int kernelBlockSize, bool analysis_flag, bool SHEMEM_flag)
{
    const unsigned int size = count * sizeof(int);// size of the input array in bytes
    const unsigned int last = count - 1;// last index of the array
    unsigned int threadCount;// number of threads to launch for each iteration
    int* dev_arr = 0;//pointer to GPU buffer for the main array
    int* dev_aux = 0;// pointer to GPU buffer for the auxiliary array
    hipError_t cudaStatus;
    double tstart, tstop, tnoshared; //tshared;
    tstart = tstop = tnoshared = 0;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    // Allocate GPU buffers for two vectors (main and aux array).
    cudaStatus = hipMalloc((void**)&dev_arr, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }
    cudaStatus = hipMalloc((void**)&dev_aux, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_arr);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_arr, arr, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }
    // chunk is Number of array elements to be sorted in single thread.
    // initially pair of elements from the array are mergeSorted in a single thread
    unsigned int chunk;

    for (chunk = 2; chunk < 2 * count; chunk *= 2) {
      // calculate the number of threads to launch based on:
      // blockSize and kernelBlockSize
      threadCount = (count / (chunk) /kernelBlockSize)+1; //added one just to make sure we have enough threads to solve the problem.

      if(threadCount>1023){
        if(analysis_flag==false){
          printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
          printf("!! Max Thread per Block is 1024        !!\n");
          printf("!! Can't fit Array size in a %d Blocks !!\n", kernelBlockSize);
          printf("!! Adjusting # Blocks per Grid ...... !!\n");
        }
        
  
        // if the number of threads is greater than the maximum limit (1024), adjust the block size
        while(threadCount>1023){
          ++kernelBlockSize;
          threadCount = (count / (chunk) /kernelBlockSize)+1;
          }
        if(analysis_flag==false){
          printf("!! Block size set to: %d              !!\n", kernelBlockSize);
          printf("!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!\n");
          }
        }

        // Define the number of blocks in the grid
        dim3 my_gridDim(kernelBlockSize);
        // Define the number of threads in each block
        dim3 my_BlockDim(threadCount);

        //Displaying the first configuration of the Threads in the GPU
        if(chunk==2 && analysis_flag==false){
          printf("First loop Configuration\n");
          printf("Block/Grid: %d \n", kernelBlockSize);
          printf("Thread/Block: %d \n", threadCount);
          printf("Total threads: %d \n", threadCount * kernelBlockSize);
          printf("T_Time: ");
        }
        if(SHEMEM_flag){
          tstart = gettime();
          // Launch the kernel with the calculated block size and number of threads
          mergeSortKernel_SHEMEM<<<my_gridDim, my_BlockDim, size*2>>>(dev_arr, dev_aux, chunk, last);
        }
        else{
          tstart = gettime();
          // Launch the kernel with the calculated block size and number of threads
          mergeSortKernel<<<my_gridDim, my_BlockDim>>>(dev_arr, dev_aux, chunk, last);
        }
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipGetLastError failed!");
            return cudaStatus;
        }
        // hipDeviceSynchronize waits for the kernel to finish, and returns
        cudaStatus = hipDeviceSynchronize();
        tstop = gettime();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize failed!");
            return cudaStatus;
        }
        tnoshared += (tstop - tstart);
    }
    printf("%f", tnoshared);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(arr, dev_arr, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        return cudaStatus;
    }

    // Free GPU buffers
    hipFree(dev_arr);
    hipFree(dev_aux);
    return hipSuccess;
}